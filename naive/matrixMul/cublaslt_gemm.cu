// cublaslt_gemm.cu
// 构建命令 (示例):
//   nvcc -O2 -std=c++17 cublaslt_gemm.cu -lcublasLt -lcublas -o cublaslt_gemm
// 可选指定架构(按你的显卡调整):
//   nvcc -O2 -std=c++17 -arch=sm_80 cublaslt_gemm.cu -lcublasLt -lcublas -o cublaslt_gemm
//
// 运行:
//   ./cublaslt_gemm [M N K]
//   例如: ./cublaslt_gemm 1024 1024 1024

#include <hip/hip_runtime.h>
#include <hipblaslt.h>

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <random>
#include <cmath>
#include <iostream>
#include <cassert>

#define CHECK_CUDA(expr) do {                               \
    hipError_t _err = (expr);                               \
    if (_err != hipSuccess) {                               \
        fprintf(stderr, "CUDA Error %s:%d: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(_err));\
        std::exit(EXIT_FAILURE);                             \
    }                                                        \
} while (0)

#define CHECK_CUBLASLT(expr) do {                            \
    hipblasStatus_t _st = (expr);                              \
    if (_st != HIPBLAS_STATUS_SUCCESS) {                      \
        fprintf(stderr, "cuBLASLt Error %s:%d: %d\n",         \
                __FILE__, __LINE__, int(_st));               \
        std::exit(EXIT_FAILURE);                             \
    }                                                        \
} while (0)

static void cpu_gemm_row_major(int M, int N, int K,
                               float alpha,
                               const float* A, int lda, // row-major: lda = K
                               const float* B, int ldb, // row-major: ldb = N
                               float beta,
                               float* C, int ldc)       // row-major: ldc = N
{
    // C = alpha * A * B + beta * C
    // 全部按 row-major 解释
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float sum = 0.f;
            for (int k = 0; k < K; ++k) {
                // A(i,k) * B(k,j)
                sum += A[i * lda + k] * B[k * ldb + j];
            }
            C[i * ldc + j] = alpha * sum + beta * C[i * ldc + j];
        }
    }
}

int main(int argc, char** argv)
{
    int M = 512, N = 512, K = 512;
    if (argc == 4) {
        M = std::atoi(argv[1]);
        N = std::atoi(argv[2]);
        K = std::atoi(argv[3]);
    }
    std::cout << "GEMM: C(" << M << "x" << N << ") = A(" << M << "x" << K
              << ") * B(" << K << "x" << N << ") [row-major, float32]\n";

    // 1) 准备主机数据（row-major）
    const int lda = K;
    const int ldb = N;
    const int ldc = N;

    std::vector<float> hA(size_t(M) * K);
    std::vector<float> hB(size_t(K) * N);
    std::vector<float> hC(size_t(M) * N);
    std::vector<float> hC_ref(size_t(M) * N);

    std::mt19937 rng(123);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    for (auto& x : hA) x = dist(rng);
    for (auto& x : hB) x = dist(rng);
    for (auto& x : hC) x = dist(rng);
    hC_ref = hC; // 保存一份给 CPU 校验

    float alpha = 1.0f, beta = 1.0f;

    // 2) 设备内存
    float *dA = nullptr, *dB = nullptr, *dC = nullptr;
    CHECK_CUDA(hipMalloc(&dA, sizeof(float) * size_t(M) * K));
    CHECK_CUDA(hipMalloc(&dB, sizeof(float) * size_t(K) * N));
    CHECK_CUDA(hipMalloc(&dC, sizeof(float) * size_t(M) * N));
    CHECK_CUDA(hipMemcpy(dA, hA.data(), sizeof(float) * size_t(M) * K, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB, hB.data(), sizeof(float) * size_t(K) * N, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dC, hC.data(), sizeof(float) * size_t(M) * N, hipMemcpyHostToDevice));

    // 3) 建立 cuBLASLt 句柄
    hipblasLtHandle_t ltHandle;
    CHECK_CUBLASLT(hipblasLtCreate(&ltHandle));

    // 4) 创建 Matmul 描述：计算类型与缩放类型
    hipblasLtMatmulDesc_t operationDesc;
    CHECK_CUBLASLT(hipblasLtMatmulDescCreate(&operationDesc,
                                            HIPBLAS_COMPUTE_32F,
                                            HIP_R_32F));

    // 设定 A/B 是否转置（此例都不转置）
    hipblasOperation_t opA = HIPBLAS_OP_N;
    hipblasOperation_t opB = HIPBLAS_OP_N;
    CHECK_CUBLASLT(hipblasLtMatmulDescSetAttribute(
            operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opA, sizeof(opA)));
    CHECK_CUBLASLT(hipblasLtMatmulDescSetAttribute(
            operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opB, sizeof(opB)));

    // 5) 创建矩阵布局（layouts），显式设置 row-major
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc;
    CHECK_CUBLASLT(hipblasLtMatrixLayoutCreate(
            &Adesc, HIP_R_32F, /*rows*/ M, /*cols*/ K, /*ld*/ lda));
    CHECK_CUBLASLT(hipblasLtMatrixLayoutCreate(
            &Bdesc, HIP_R_32F, /*rows*/ K, /*cols*/ N, /*ld*/ ldb));
    CHECK_CUBLASLT(hipblasLtMatrixLayoutCreate(
            &Cdesc, HIP_R_32F, /*rows*/ M, /*cols*/ N, /*ld*/ ldc));

    hipblasLtOrder_t order = HIPBLASLT_ORDER_ROW; // 行优先
    CHECK_CUBLASLT(hipblasLtMatrixLayoutSetAttribute(
            Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));
    CHECK_CUBLASLT(hipblasLtMatrixLayoutSetAttribute(
            Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));
    CHECK_CUBLASLT(hipblasLtMatrixLayoutSetAttribute(
            Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));

    // 6) (可选) workspace + heuristic 选算法
    size_t workspaceSize = 32ull << 20; // 32 MB
    void* dWorkspace = nullptr;
    CHECK_CUDA(hipMalloc(&dWorkspace, workspaceSize));

    hipblasLtMatmulPreference_t preference;
    CHECK_CUBLASLT(hipblasLtMatmulPreferenceCreate(&preference));
    CHECK_CUBLASLT(hipblasLtMatmulPreferenceSetAttribute(
            preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
            &workspaceSize, sizeof(workspaceSize)));

    hipblasLtMatmulHeuristicResult_t heuristicResult;
    int returnedResults = 0;
    CHECK_CUBLASLT(hipblasLtMatmulAlgoGetHeuristic(
            ltHandle,
            operationDesc,
            Adesc,
            Bdesc,
            Cdesc,
            Cdesc,
            preference,
            1, // 请求一个可用算法
            &heuristicResult,
            &returnedResults));

    if (returnedResults == 0) {
        fprintf(stderr, "No suitable cuBLASLt Matmul algorithm found.\n");
        std::exit(EXIT_FAILURE);
    }

    // 7) 执行 Matmul
    hipStream_t stream = 0; // 用默认流
    CHECK_CUBLASLT(hipblasLtMatmul(
            ltHandle,
            operationDesc,
            &alpha,
            dA, Adesc,
            dB, Bdesc,
            &beta,
            dC, Cdesc,
            dC, Cdesc,
            &heuristicResult.algo,
            dWorkspace, workspaceSize,
            stream));

    // 等待计算完成
    CHECK_CUDA(hipDeviceSynchronize());

    // 8) 拷回并用 CPU 校验
    CHECK_CUDA(hipMemcpy(hC.data(), dC, sizeof(float) * size_t(M) * N, hipMemcpyDeviceToHost));

    cpu_gemm_row_major(M, N, K, alpha, hA.data(), lda, hB.data(), ldb, beta, hC_ref.data(), ldc);

    // 计算误差
    double max_abs_err = 0.0, max_rel_err = 0.0;
    for (size_t i = 0; i < hC.size(); ++i) {
        double ref = hC_ref[i];
        double got = hC[i];
        double abs_err = std::abs(ref - got);
        double rel_err = abs_err / (std::abs(ref) + 1e-8);
        if (abs_err > max_abs_err) max_abs_err = abs_err;
        if (rel_err > max_rel_err) max_rel_err = rel_err;
    }
    std::cout << "Max abs err: " << max_abs_err
              << " | Max rel err: " << max_rel_err << "\n";

    // 9) 资源清理
    CHECK_CUDA(hipFree(dWorkspace));
    CHECK_CUBLASLT(hipblasLtMatmulPreferenceDestroy(preference));

    CHECK_CUBLASLT(hipblasLtMatrixLayoutDestroy(Adesc));
    CHECK_CUBLASLT(hipblasLtMatrixLayoutDestroy(Bdesc));
    CHECK_CUBLASLT(hipblasLtMatrixLayoutDestroy(Cdesc));
    CHECK_CUBLASLT(hipblasLtMatmulDescDestroy(operationDesc));
    CHECK_CUBLASLT(hipblasLtDestroy(ltHandle));

    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(dC));

    std::cout << "Done.\n";
    return 0;
}
